#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include "md5.h"
//630188182
#define SIZE 630188182 
#define MAX_LENGTH 7 
#define CHARSETLEN 26 


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__device__ __constant__ unsigned char cudaCharSet[95];
__device__ unsigned char correctPass[MAX_LENGTH];

__global__ void bruteForce(unsigned char *prefixes, int length, int offset_start, int offeset_end, int prefix_to_calc, uint v1, uint v2, uint v3, uint v4, unsigned char *password_d)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int local_threadId = threadIdx.x + threadIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    //prefixes[offset + threadId + length - 1] = 'a';

    if (blockId < prefix_to_calc - 1 && local_threadId<CHARSETLEN-1) {
        //Need to optimize 7758650592 
        memcpy(prefixes + (offeset_end  + threadId)*MAX_LENGTH, prefixes+(offset_start+blockId)*MAX_LENGTH, (length - 1) * sizeof(char));
        prefixes[(offeset_end + threadId) * MAX_LENGTH + length - 1] = cudaCharSet[local_threadId];
        uint c1 = 0, c2 = 0, c3 = 0, c4 = 0;
        md5_vfy(prefixes + (offeset_end + threadId) * MAX_LENGTH, length, &c1, &c2, &c3, &c4);
        if (c1 == v1 && c2 == v2 && c3 == v3 && c4 == v4)
        {
            memcpy(password_d, prefixes + (offeset_end + threadId) * MAX_LENGTH, length  * sizeof(char));
        }
    }
}

int main(int argc, unsigned char* argv[])
{
    char charset[] = "abcdefghijklmnopqrstuvwxyz";
    //af94ffbb0e815172b1160d4b58a3ece3 -> imanol
    //172346606e1d24062e891d537e917a90 -> lolol 
    unsigned char hash[33] = "a097897098930ad07bf6db97a8d10b83";
    unsigned char password[MAX_LENGTH], * password_d;

    uint v1, v2, v3, v4;
    md5_to_ints(argv[1], &v1, &v2, &v3, &v4);
    //  printf("%i %i %i %i", v1, v2, v3, v4);
    hipMemcpyToSymbol(HIP_SYMBOL(cudaCharSet), &charset, CHARSETLEN, 0, hipMemcpyHostToDevice);
    long prefix_n = 26;
    int length = 5;
    long offset_start = 0;
    long offest_end = 26;
    unsigned char* prefixes_d, * prefixes;
    prefixes = (unsigned char*)malloc(SIZE * sizeof(char) * MAX_LENGTH);
    memset(prefixes, '\0', SIZE * sizeof(unsigned char) * MAX_LENGTH);
    //load first prefix
    for (int i = 0; i < CHARSETLEN; i++) prefixes[i * MAX_LENGTH] = charset[i];
    hipMalloc((void**)&prefixes_d, SIZE * sizeof(unsigned char) * MAX_LENGTH);
    hipMalloc((void**)&password_d, MAX_LENGTH * sizeof(unsigned char) );

    hipMemcpy(prefixes_d, prefixes, CHARSETLEN * MAX_LENGTH * sizeof(unsigned char), hipMemcpyHostToDevice);

    for (int i = 2; i <= length; i++) {
        long dim = (int)ceil(sqrt((float)prefix_n));
        printf("dim: %llu\n", dim);

        dim3 grid_dim(dim, dim);
        printf("prefix: %llu\n", prefix_n);
        prefix_n = prefix_n* CHARSETLEN;
        printf("prefix: %llu\n", prefix_n);
        bruteForce <<< grid_dim, 26 >> > (prefixes_d, i, offset_start, offest_end, prefix_n, v1, v2, v3, v4,password_d);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        
        hipMemcpy(password, password_d, MAX_LENGTH * sizeof(unsigned char), hipMemcpyDeviceToHost);
        if (password[0] != '\0')
        {
            printf("We found %s as %s  ", hash, password);
            break;
        }
        

     //   printf("offset_start: %llu\n", offset_start);
        offset_start = offest_end;
     //   printf("offset_start: %llu\n", offset_start);
     //   printf("offset_end: %llu\n", offest_end);
        offest_end = offset_start + prefix_n;
     //   printf("offest_end: %llu\n", offest_end);

    }

    hipMemcpy(prefixes, prefixes_d, SIZE * sizeof(unsigned char) * MAX_LENGTH, hipMemcpyDeviceToHost);

  /** for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < 5; j++)printf("%c", prefixes[i * MAX_LENGTH + j]);
            printf("\n");
    }**/

   /** for (int j = 0; j < 5; j++)printf("%c", prefixes[475254 * MAX_LENGTH + j]);
    printf("\n");*/

}
